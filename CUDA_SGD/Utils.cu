#include "Utils.cuh"
#include<stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>



/********************/
/* CUDA ERROR CHECK */
/********************/
void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) { exit(code); }
	}
}


/*******************/
/* iDivUp FUNCTION */
/*******************/
//extern "C" int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }
__host__ __device__ int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }


void gpuErrchk(hipError_t ans) { gpuAssert((ans), __FILE__, __LINE__); }


